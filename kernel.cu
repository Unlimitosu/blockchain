#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__device__ static unsigned int cuda_keccakRate = 0;
__device__ static unsigned int cuda_keccakCapacity = 0;
__device__ static unsigned int cuda_keccakSuffix = 0;
__device__ static int cuda_end_offset = 0;

static unsigned int keccakRate = 0;
static unsigned int keccakCapacity = 0;
static unsigned int keccakSuffix = 0;
static int end_offset = 0;

__constant__ static const uint32_t cuda_keccakf_rndc[KECCAK_ROUND][2] =
{
	{0x00000001, 0x00000000}, {0x00008082, 0x00000000},
	{0x0000808a, 0x80000000}, {0x80008000, 0x80000000},
	{0x0000808b, 0x00000000}, {0x80000001, 0x00000000},
	{0x80008081, 0x80000000}, {0x00008009, 0x80000000},
	{0x0000008a, 0x00000000}, {0x00000088, 0x00000000},
	{0x80008009, 0x00000000}, {0x8000000a, 0x00000000},

	{0x8000808b, 0x00000000}, {0x0000008b, 0x80000000},
	{0x00008089, 0x80000000}, {0x00008003, 0x80000000},
	{0x00008002, 0x80000000}, {0x00000080, 0x80000000},
	{0x0000800a, 0x00000000}, {0x8000000a, 0x80000000},
	{0x80008081, 0x80000000}, {0x00008080, 0x80000000},
	{0x80000001, 0x00000000}, {0x80008008, 0x80000000}
};
__constant__ unsigned int cuda_keccakf_rotc[KECCAK_ROUND] =
{
	 1,  3,  6, 10, 15, 21, 28, 36, 45, 55,  2, 14,
	27, 41, 56,  8, 25, 43, 62, 18, 39, 61, 20, 44
};
__constant__ unsigned int cuda_keccakf_piln[KECCAK_ROUND] =
{
	10,  7, 11, 17, 18,  3,  5, 16,  8, 21, 24,  4,
	15, 23, 19, 13, 12,  2, 20, 14, 22,  9,  6,  1
};

static const uint32_t keccakf_rndc[KECCAK_ROUND][2] =
{
	{0x00000001, 0x00000000}, {0x00008082, 0x00000000},
	{0x0000808a, 0x80000000}, {0x80008000, 0x80000000},
	{0x0000808b, 0x00000000}, {0x80000001, 0x00000000},
	{0x80008081, 0x80000000}, {0x00008009, 0x80000000},
	{0x0000008a, 0x00000000}, {0x00000088, 0x00000000},
	{0x80008009, 0x00000000}, {0x8000000a, 0x00000000},

	{0x8000808b, 0x00000000}, {0x0000008b, 0x80000000},
	{0x00008089, 0x80000000}, {0x00008003, 0x80000000},
	{0x00008002, 0x80000000}, {0x00000080, 0x80000000},
	{0x0000800a, 0x00000000}, {0x8000000a, 0x80000000},
	{0x80008081, 0x80000000}, {0x00008080, 0x80000000},
	{0x80000001, 0x00000000}, {0x80008008, 0x80000000}
};
unsigned int keccakf_rotc[KECCAK_ROUND] =
{
	 1,  3,  6, 10, 15, 21, 28, 36, 45, 55,  2, 14,
	27, 41, 56,  8, 25, 43, 62, 18, 39, 61, 20, 44
};
unsigned int keccakf_piln[KECCAK_ROUND] =
{
	10,  7, 11, 17, 18,  3,  5, 16,  8, 21, 24,  4,
	15, 23, 19, 13, 12,  2, 20, 14, 22,  9,  6,  1
};


__device__ void cuda_ROL64(uint32_t* in, uint32_t* out, unsigned int offset)
{
	int shift = 0;

	if (offset == 0)
	{
		out[1] = in[1];
		out[0] = in[0];
	}
	else if (offset < 32)
	{
		shift = offset;

		out[1] = (uint32_t)((in[1] << shift) ^ (in[0] >> (32 - shift)));
		out[0] = (uint32_t)((in[0] << shift) ^ (in[1] >> (32 - shift)));
	}
	else if (offset < 64)
	{
		shift = offset - 32;

		out[1] = (uint32_t)((in[0] << shift) ^ (in[1] >> (32 - shift)));
		out[0] = (uint32_t)((in[1] << shift) ^ (in[0] >> (32 - shift)));
	}
	else
	{
		out[1] = in[1];
		out[0] = in[0];
	}
}
void ROL64(uint32_t* in, uint32_t* out, unsigned int offset)
{
	int shift = 0;

	if (offset == 0)
	{
		out[1] = in[1];
		out[0] = in[0];
	}
	else if (offset < 32)
	{
		shift = offset;

		out[1] = (uint32_t)((in[1] << shift) ^ (in[0] >> (32 - shift)));
		out[0] = (uint32_t)((in[0] << shift) ^ (in[1] >> (32 - shift)));
	}
	else if (offset < 64)
	{
		shift = offset - 32;

		out[1] = (uint32_t)((in[0] << shift) ^ (in[1] >> (32 - shift)));
		out[0] = (uint32_t)((in[1] << shift) ^ (in[0] >> (32 - shift)));
	}
	else
	{
		out[1] = in[1];
		out[0] = in[0];
	}
}

__device__ void cuda_keccakf(uint8_t* state)
{
	uint32_t t[2], bc[5][2], s[25][2] = { 0x00, };
	int i, j, round;

	for (i = 0; i < 25; i++)
	{
		s[i][0] = (uint32_t)(state[i * 8 + 0]) |
			(uint32_t)(state[i * 8 + 1] << 8) |
			(uint32_t)(state[i * 8 + 2] << 16) |
			(uint32_t)(state[i * 8 + 3] << 24);
		s[i][1] = (uint32_t)(state[i * 8 + 4]) |
			(uint32_t)(state[i * 8 + 5] << 8) |
			(uint32_t)(state[i * 8 + 6] << 16) |
			(uint32_t)(state[i * 8 + 7] << 24);
	}

	for (round = 0; round < KECCAK_ROUND; round++)
	{
		/* Theta */
		for (i = 0; i < 5; i++)
		{
			bc[i][0] = s[i][0] ^ s[i + 5][0] ^ s[i + 10][0] ^ s[i + 15][0] ^ s[i + 20][0];
			bc[i][1] = s[i][1] ^ s[i + 5][1] ^ s[i + 10][1] ^ s[i + 15][1] ^ s[i + 20][1];
		}

		for (i = 0; i < 5; i++)
		{
			cuda_ROL64(bc[(i + 1) % 5], t, 1);

			t[0] ^= bc[(i + 4) % 5][0];
			t[1] ^= bc[(i + 4) % 5][1];

			for (j = 0; j < 25; j += 5)
			{
				s[j + i][0] ^= t[0];
				s[j + i][1] ^= t[1];
			}
		}

		/* Rho & Pi */
		t[0] = s[1][0];
		t[1] = s[1][1];

		for (i = 0; i < KECCAK_ROUND; i++)
		{
			j = cuda_keccakf_piln[i];

			bc[0][0] = s[j][0];
			bc[0][1] = s[j][1];

			cuda_ROL64(t, s[j], cuda_keccakf_rotc[i]);

			t[0] = bc[0][0];
			t[1] = bc[0][1];
		}

		/* Chi */
		for (j = 0; j < 25; j += 5)
		{
			for (i = 0; i < 5; i++)
			{
				bc[i][0] = s[j + i][0];
				bc[i][1] = s[j + i][1];
			}

			for (i = 0; i < 5; i++)
			{
				s[j + i][0] ^= (~bc[(i + 1) % 5][0]) & bc[(i + 2) % 5][0];
				s[j + i][1] ^= (~bc[(i + 1) % 5][1]) & bc[(i + 2) % 5][1];
			}
		}

		/* Iota */
		s[0][0] ^= cuda_keccakf_rndc[round][0];
		s[0][1] ^= cuda_keccakf_rndc[round][1];
	}

	for (i = 0; i < 25; i++)
	{
		state[i * 8 + 0] = (uint8_t)(s[i][0]);
		state[i * 8 + 1] = (uint8_t)(s[i][0] >> 8);
		state[i * 8 + 2] = (uint8_t)(s[i][0] >> 16);
		state[i * 8 + 3] = (uint8_t)(s[i][0] >> 24);
		state[i * 8 + 4] = (uint8_t)(s[i][1]);
		state[i * 8 + 5] = (uint8_t)(s[i][1] >> 8);
		state[i * 8 + 6] = (uint8_t)(s[i][1] >> 16);
		state[i * 8 + 7] = (uint8_t)(s[i][1] >> 24);
	}
}
void keccakf(uint8_t* state)
{
	uint32_t t[2], bc[5][2], s[25][2] = { 0x00, };
	int i, j, round;
	for (i = 0; i < 25; i++)
	{
		s[i][0] = (uint32_t)(state[i * 8 + 0]) |
			(uint32_t)(state[i * 8 + 1] << 8) |
			(uint32_t)(state[i * 8 + 2] << 16) |
			(uint32_t)(state[i * 8 + 3] << 24);
		s[i][1] = (uint32_t)(state[i * 8 + 4]) |
			(uint32_t)(state[i * 8 + 5] << 8) |
			(uint32_t)(state[i * 8 + 6] << 16) |
			(uint32_t)(state[i * 8 + 7] << 24);
	}

	for (round = 0; round < KECCAK_ROUND; round++)
	{
		/* Theta */
		for (i = 0; i < 5; i++)
		{
			bc[i][0] = s[i][0] ^ s[i + 5][0] ^ s[i + 10][0] ^ s[i + 15][0] ^ s[i + 20][0];
			bc[i][1] = s[i][1] ^ s[i + 5][1] ^ s[i + 10][1] ^ s[i + 15][1] ^ s[i + 20][1];
		}

		for (i = 0; i < 5; i++)
		{
			ROL64(bc[(i + 1) % 5], t, 1);

			t[0] ^= bc[(i + 4) % 5][0];
			t[1] ^= bc[(i + 4) % 5][1];

			for (j = 0; j < 25; j += 5)
			{
				s[j + i][0] ^= t[0];
				s[j + i][1] ^= t[1];
			}
		}

		/* Rho & Pi */
		t[0] = s[1][0];
		t[1] = s[1][1];

		for (i = 0; i < KECCAK_ROUND; i++)
		{
			j = keccakf_piln[i];

			bc[0][0] = s[j][0];
			bc[0][1] = s[j][1];

			ROL64(t, s[j], keccakf_rotc[i]);

			t[0] = bc[0][0];
			t[1] = bc[0][1];
		}

		/* Chi */
		for (j = 0; j < 25; j += 5)
		{
			for (i = 0; i < 5; i++)
			{
				bc[i][0] = s[j + i][0];
				bc[i][1] = s[j + i][1];
			}

			for (i = 0; i < 5; i++)
			{
				s[j + i][0] ^= (~bc[(i + 1) % 5][0]) & bc[(i + 2) % 5][0];
				s[j + i][1] ^= (~bc[(i + 1) % 5][1]) & bc[(i + 2) % 5][1];
			}
		}

		/* Iota */
		s[0][0] ^= keccakf_rndc[round][0];
		s[0][1] ^= keccakf_rndc[round][1];
	}

	for (i = 0; i < 25; i++)
	{
		state[i * 8 + 0] = (uint8_t)(s[i][0]);
		state[i * 8 + 1] = (uint8_t)(s[i][0] >> 8);
		state[i * 8 + 2] = (uint8_t)(s[i][0] >> 16);
		state[i * 8 + 3] = (uint8_t)(s[i][0] >> 24);
		state[i * 8 + 4] = (uint8_t)(s[i][1]);
		state[i * 8 + 5] = (uint8_t)(s[i][1] >> 8);
		state[i * 8 + 6] = (uint8_t)(s[i][1] >> 16);
		state[i * 8 + 7] = (uint8_t)(s[i][1] >> 24);
	}
}

__device__ int cuda_keccak_absorb(uint8_t* input, int inLen, int rate, int capacity, uint8_t* keccak_state)
{
	uint8_t* buf = input;
	int iLen = inLen;
	int rateInBytes = rate / 8;
	int blockSize = 0;
	int i = 0;

	if ((rate + capacity) != KECCAK_SPONGE_BIT)
		return SHA3_PARAMETER_ERROR;

	if (((rate % 8) != 0) || (rate < 1))
		return SHA3_PARAMETER_ERROR;

	while (iLen > 0)
	{
		if ((cuda_end_offset != 0) && (cuda_end_offset < rateInBytes))
		{
			blockSize = (((iLen + cuda_end_offset) < rateInBytes) ? (iLen + cuda_end_offset) : rateInBytes);

			for (i = cuda_end_offset; i < blockSize; i++)
				keccak_state[i] ^= buf[i - cuda_end_offset];

			buf += blockSize - cuda_end_offset;
			iLen -= blockSize - cuda_end_offset;
		}
		else
		{
			blockSize = ((iLen < rateInBytes) ? iLen : rateInBytes);

			for (i = 0; i < blockSize; i++)
				keccak_state[i] ^= buf[i];

			buf += blockSize;
			iLen -= blockSize;
		}

		if (blockSize == rateInBytes)
		{
			cuda_keccakf(keccak_state);
			blockSize = 0;
		}

		cuda_end_offset = blockSize;
	}

	return SHA3_OK;
}
int keccak_absorb(uint8_t* input, int inLen, int rate, int capacity, uint8_t* keccak_state)
{
	uint8_t* buf = input;
	int iLen = inLen;
	int rateInBytes = rate / 8;
	int blockSize = 0;
	int i = 0;

	if ((rate + capacity) != KECCAK_SPONGE_BIT)
		return SHA3_PARAMETER_ERROR;

	if (((rate % 8) != 0) || (rate < 1))
		return SHA3_PARAMETER_ERROR;

	while (iLen > 0)
	{
		if ((end_offset != 0) && (end_offset < rateInBytes))
		{
			blockSize = (((iLen + end_offset) < rateInBytes) ? (iLen + end_offset) : rateInBytes);

			for (i = end_offset; i < blockSize; i++)
				keccak_state[i] ^= buf[i - end_offset];

			buf += blockSize - end_offset;
			iLen -= blockSize - end_offset;
		}
		else
		{
			blockSize = ((iLen < rateInBytes) ? iLen : rateInBytes);

			for (i = 0; i < blockSize; i++)
				keccak_state[i] ^= buf[i];

			buf += blockSize;
			iLen -= blockSize;
		}

		if (blockSize == rateInBytes)
		{
			keccakf(keccak_state);
			blockSize = 0;
		}

		end_offset = blockSize;
	}

	return SHA3_OK;
}

__device__ void __cuda_sha3_init(int bitSize, int useSHAKE, uint8_t* keccak_state)
{
	cuda_keccakCapacity = bitSize * 2;
	cuda_keccakRate = KECCAK_SPONGE_BIT - cuda_keccakCapacity;
	if (useSHAKE)
		cuda_keccakSuffix = KECCAK_SHAKE_SUFFIX;
	else
		cuda_keccakSuffix = KECCAK_SHA3_SUFFIX;
	memset(keccak_state, 0x00, KECCAK_STATE_SIZE);
	cuda_end_offset = 0;
}
void __sha3_init_endoffset16(int bitSize, int useSHAKE, uint8_t* keccak_state)
{
	keccakCapacity = bitSize * 2;
	keccakRate = KECCAK_SPONGE_BIT - keccakCapacity;
	if (useSHAKE)
		keccakSuffix = KECCAK_SHAKE_SUFFIX;
	else
		keccakSuffix = KECCAK_SHA3_SUFFIX;
	//memset(keccak_state, 0x00, KECCAK_STATE_SIZE);
	end_offset = 16;
}
void __sha3_init(int bitSize, int useSHAKE, uint8_t* keccak_state)
{
	keccakCapacity = bitSize * 2;
	keccakRate = KECCAK_SPONGE_BIT - keccakCapacity;
	if (useSHAKE)
		keccakSuffix = KECCAK_SHAKE_SUFFIX;
	else
		keccakSuffix = KECCAK_SHA3_SUFFIX;
	//memset(keccak_state, 0x00, KECCAK_STATE_SIZE);
	end_offset = 0;
}

__device__ void cuda_sha3_update(uint8_t* input, int inLen, uint8_t* keccak_state)
{
	cuda_keccak_absorb(input, inLen, cuda_keccakRate, cuda_keccakCapacity, keccak_state);
}
void sha3_update(uint8_t* input, int inLen, uint8_t* keccak_state)
{
	keccak_absorb(input, inLen, keccakRate, keccakCapacity, keccak_state);
}

__device__ void cuda_sha3_hash(uint8_t* input, int inLen, int bitSize, uint8_t* keccak_state)
{
	cuda_sha3_update(input, inLen, keccak_state);
}
void sha3_hash(uint8_t* input, int inLen, int bitSize, uint8_t* keccak_state)
{
	sha3_update(input, inLen, keccak_state);
}

__device__ void cuda_sha3_init(int bitSize, uint8_t* keccak_state)
{
	__cuda_sha3_init(bitSize, SHA3_SHAKE_NONE, keccak_state);
}
void sha3_init_endoffset16(int bitSize, uint8_t* keccak_state)
{
	__sha3_init_endoffset16(bitSize, SHA3_SHAKE_NONE, keccak_state);
}
void sha3_init(int bitSize, uint8_t* keccak_state)
{
	__sha3_init(bitSize, SHA3_SHAKE_NONE, keccak_state);
}
//ㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡ
// 
//CUDA verify section
__device__ void cuda_verify_sha3_init(int bitSize, uint8_t* keccak_state) {
	cuda_keccakCapacity = bitSize * 2;
	cuda_keccakRate = KECCAK_SPONGE_BIT - cuda_keccakCapacity;
	if (SHA3_SHAKE_NONE)
		cuda_keccakSuffix = KECCAK_SHAKE_SUFFIX;
	else
		cuda_keccakSuffix = KECCAK_SHA3_SUFFIX;
	cuda_end_offset = 0;
}
__device__ int cuda_verify_sha3_update(uint8_t* input, int inLen, uint8_t* keccak_state) {
	uint8_t* buf = input;
	int iLen = inLen;
	int rateInBytes = cuda_keccakRate / 8;
	int blockSize = 0;
	int i = 0;

	if ((cuda_keccakRate + cuda_keccakCapacity) != KECCAK_SPONGE_BIT)
		return SHA3_PARAMETER_ERROR;

	if (((cuda_keccakRate % 8) != 0) || (cuda_keccakRate < 1))
		return SHA3_PARAMETER_ERROR;

	while (iLen > 0)
	{
		if ((cuda_end_offset != 0) && (cuda_end_offset < rateInBytes))
		{
			blockSize = (((iLen + cuda_end_offset) < rateInBytes) ? (iLen + cuda_end_offset) : rateInBytes);

			for (i = cuda_end_offset; i < blockSize; i++)
				keccak_state[i] ^= buf[i - cuda_end_offset];

			buf += blockSize - cuda_end_offset;
			iLen -= blockSize - cuda_end_offset;
		}
		else
		{
			blockSize = ((iLen < rateInBytes) ? iLen : rateInBytes);

			for (i = 0; i < blockSize; i++)
				keccak_state[i] ^= buf[i];

			buf += blockSize;
			iLen -= blockSize;
		}

		if (blockSize == rateInBytes)
		{
			cuda_keccakf(keccak_state);
			blockSize = 0;
		}

		cuda_end_offset = blockSize;
	}

	return SHA3_OK;
}

__device__ int cuda_verify_sha3_final(uint8_t* output, int outLen, int rate, int suffix, uint8_t* keccak_state)
{
	uint8_t* buf = output;
	int oLen = outLen;
	int rateInBytes = rate / 8;
	int blockSize = cuda_end_offset;
	int i = 0;

	keccak_state[blockSize] ^= suffix;

	if (((suffix & 0x80) != 0) && (blockSize == (rateInBytes - 1)))
		cuda_keccakf(keccak_state);

	keccak_state[rateInBytes - 1] ^= 0x80;

	cuda_keccakf(keccak_state);

	while (oLen > 0)
	{
		blockSize = ((oLen < rateInBytes) ? oLen : rateInBytes);
		for (i = 0; i < blockSize; i++)
			buf[i] = keccak_state[i];
		buf += blockSize;
		oLen -= blockSize;

		if (oLen > 0)
			cuda_keccakf(keccak_state);
	}

	return SHA3_OK;
}

//ㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡㅡ
__global__ void cuda_sha3(uint8_t* data, uint8_t* dev_keccak_state)
{
	uint8_t in[1024 * 8] = { 0, }; //124
	int in_length = 1024 * 8;	//byte size
	int hash_bit = 256;		//bit(224,256,384,512)
	int index = 0;	// 각 thread index
	uint8_t keccak_state[KECCAK_STATE_SIZE] = { 0x00, };

	cuda_sha3_init(hash_bit, keccak_state);

	for (int i = 0; i < 1024 / 8; i++) {
		index = (1024 * 1024 * blockIdx.x * blockDim.x + 1024 * 1024 * threadIdx.x) + (i * 1024 * 8);
		for (int j = 0; j < 1024 * 8; j++) 
			in[j] = data[index++];	
		cuda_sha3_hash(in, in_length, hash_bit, keccak_state);
	}
	index = KECCAK_STATE_SIZE * blockIdx.x * blockDim.x + (KECCAK_STATE_SIZE * threadIdx.x);

	for (int i = 0; i < KECCAK_STATE_SIZE; i++) 
		dev_keccak_state[index++] = keccak_state[i];
	/*if (threadIdx.x == 0) {
		printf("threadIdx = 0\n");
		for (int i = 0; i < KECCAK_STATE_SIZE; i++) {
			printf("%02X ", keccak_state[i]);
			if ((i + 1) % 32 == 0)
				printf("\n");
		}
		printf("\n");
	}
	__syncthreads();
	if (threadIdx.x == 1) {
		printf("threadIdx.x = 1\n");
		for (int i = 0; i < KECCAK_STATE_SIZE; i++) {
			printf("%02X ", keccak_state[i]);
			if ((i + 1) % 32 == 0)
				printf("\n");
		}
		printf("\n");
	}
	__syncthreads();
	if (threadIdx.x == 2) {
		printf("threadIdx.x = 2\n");
		for (int i = 0; i < KECCAK_STATE_SIZE; i++) {
			printf("%02X ", keccak_state[i]);
			if ((i + 1) % 32 == 0)
				printf("\n");
		}
		printf("\n");
	}
	__syncthreads();
	if (threadIdx.x == 3) {
		printf("threadIdx.x = 3\n");
		for (int i = 0; i < KECCAK_STATE_SIZE; i++) {
			printf("%02X ", keccak_state[i]);
			if ((i + 1) % 32 == 0)
				printf("\n");
		}
		printf("\n");
	}
	__syncthreads();
	if (threadIdx.x == 4) {
		printf("threadIdx.x = 4\n");
		for (int i = 0; i < KECCAK_STATE_SIZE; i++) {
			printf("%02X ", keccak_state[i]);
			if ((i + 1) % 32 == 0)
				printf("\n");
		}
		printf("\n");
	}
	__syncthreads();*/

}

int keccak_squeeze(uint8_t* output, int outLen, int rate, int suffix, uint8_t* keccak_state)
{
	uint8_t* buf = output;
	int oLen = outLen;
	int rateInBytes = rate / 8;
	int blockSize = end_offset;
	int i = 0;

	keccak_state[blockSize] ^= suffix;

	if (((suffix & 0x80) != 0) && (blockSize == (rateInBytes - 1)))
		keccakf(keccak_state);

	keccak_state[rateInBytes - 1] ^= 0x80;

	keccakf(keccak_state);

	while (oLen > 0)
	{
		blockSize = ((oLen < rateInBytes) ? oLen : rateInBytes);
		for (i = 0; i < blockSize; i++)
			buf[i] = keccak_state[i];
		buf += blockSize;
		oLen -= blockSize;

		if (oLen > 0)
			keccakf(keccak_state);
	}

	return SHA3_OK;
}

void sha3_final(uint8_t* output, uint8_t* keccak_state)
{

	keccak_squeeze(output, 32, keccakRate, keccakSuffix, keccak_state);
	keccakRate = 0;
	keccakCapacity = 0;
	keccakSuffix = 0;
}


void sha3_endoffset16(uint8_t* data, int datalen, uint8_t* out, uint8_t* keccak_state) {
	uint8_t in[1024] = { 0, };
	int in_length = 1024 * 8;	//byte size
	int hash_bit = 256;		//bit(224,256,384,512)
	int index = 0;	// 각 thread index

	sha3_init_endoffset16(hash_bit, keccak_state);
	sha3_update(data, datalen, keccak_state);
}
void sha3(uint8_t* data, int datalen, uint8_t* out, uint8_t* keccak_state) {
	uint8_t in[1024] = { 0, };
	int in_length = 1024 * 8;	//byte size
	int hash_bit = 256;		//bit(224,256,384,512)
	int index = 0;	// 각 thread index

	sha3_init(hash_bit, keccak_state);
	sha3_update(data, datalen, keccak_state);
	//sha3_final(out, keccak_state);
}
void sha3_cpu_to_gpu(uint8_t * data, uint8_t * keccak_state, int message_num){
	uint8_t* dev_data = NULL;
	uint8_t* dev_keccak_state = NULL;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_data, 1024 * 1024 * message_num * sizeof(uint8_t));
	cudaStatus = hipMalloc((void**)&dev_keccak_state, KECCAK_STATE_SIZE * message_num * sizeof(uint8_t));

	cudaStatus = hipMemcpy(dev_data, data, 1024 * 1024 * message_num * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("dev_data Error\n");
		return;
	}
	cudaStatus = hipMemcpy(dev_keccak_state, keccak_state, KECCAK_STATE_SIZE * message_num * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("dev_keccak_state Error\n");
		return;
	}
	if (message_num == MESSAGE_NUM) {
		cuda_sha3 << < GPB, BPT >> > (dev_data, dev_keccak_state);
	}
	else {
		int gpb = message_num / 192;
		int bpt = message_num % 192;
		if (gpb) {
			cuda_sha3 << < gpb, BPT >> > (dev_data, dev_keccak_state);
		}
		if (bpt) {
			cuda_sha3 << < 1, bpt >> > (dev_data + gpb * BPT * 1024 * 1024, dev_keccak_state + gpb * BPT * KECCAK_STATE_SIZE);
		}
	}
	cudaStatus = hipGetLastError();
	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(keccak_state, dev_keccak_state, KECCAK_STATE_SIZE * message_num * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(dev_data);
	hipFree(dev_keccak_state);
}

//! 
__global__ void verify_gen_hash(uint8_t* TX, uint8_t* bid, uint8_t* prehash, uint8_t* dev_hash)
{
	uint8_t in[(1024 * 8)] = { 0, }; //124
	int in_length = 1024 * 8;	//byte size
	int hash_bit = 256;		//bit(224,256,384,512)
	int index = 0;	// 각 thread index
	uint8_t keccak_state[KECCAK_STATE_SIZE] = { 0x00, };
	
	cuda_verify_sha3_init(hash_bit, keccak_state);
	for (int i = 0; i < 1024 / 8; i++) {
		index = (1024 * 1024 * blockIdx.x * blockDim.x + 1024 * 1024 * threadIdx.x) + (i * 1024 * 8);
		for (int j = 0; j < 1024 * 8; j++)
			in[j] = TX[index++];
		cuda_verify_sha3_update(in, in_length, keccak_state);
	}
	//! 마지막 40-byte 처리
	index = (20 * blockIdx.x * blockDim.x + 20 * threadIdx.x);
	for (int j = 0; j < 20; j++) {
		in[j] = bid[index + j];
		in[j + 20] = prehash[index + j];
	}
	cuda_verify_sha3_update(in, 40, keccak_state);
	index = (32 * blockIdx.x * blockDim.x + 32 * threadIdx.x);
	cuda_verify_sha3_final(dev_hash + index, 32, cuda_keccakRate, cuda_keccakSuffix, keccak_state);
}

void sha3_verify_cpu_to_gpu(uint8_t* TX, uint8_t* bid, uint8_t* prehash, uint8_t* out, int message_num) {
	uint8_t* dev_TX = NULL;
	uint8_t* dev_bid = NULL;
	uint8_t* dev_prehash = NULL;
	uint8_t* dev_out = NULL;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_TX, 1024 * 1024 * message_num * sizeof(uint8_t));
	cudaStatus = hipMalloc((void**)&dev_bid, 20 * message_num * sizeof(uint8_t));
	cudaStatus = hipMalloc((void**)&dev_prehash, 20 * message_num * sizeof(uint8_t));
	cudaStatus = hipMalloc((void**)&dev_out, 32 * message_num * sizeof(uint8_t));


	cudaStatus = hipMemcpy(dev_TX, TX, 1024 * 1024 * message_num * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("dev_data Error\n");
		return;
	}
	cudaStatus = hipMemcpy(dev_bid, bid, 20 * message_num * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("dev_bid Error\n");
		return;
	}
	cudaStatus = hipMemcpy(dev_prehash, prehash, 20 * message_num * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("dev_prehash Error\n");
		return;
	}

	if (message_num == MESSAGE_NUM) {
		verify_gen_hash << <GPB, BPT >> > (dev_TX, dev_bid, dev_prehash, dev_out);
		//cuda_sha3 << < GPB, BPT >> > (dev_data, dev_keccak_state);
	}
	else {
		int gpb = message_num / 192;
		int bpt = message_num % 192;
		if (gpb) {
			verify_gen_hash << < gpb, BPT >> > (dev_TX, dev_bid, dev_prehash, dev_out);
		}
		if (bpt) {
			//verify_gen_hash << < 1, bpt >> > (dev_data + gpb * BPT * 1024 * 1024, dev_keccak_state + gpb * BPT * KECCAK_STATE_SIZE);
			verify_gen_hash << < 1, bpt >> > (dev_TX + gpb * BPT * 1024 * 1024, dev_bid + gpb * BPT * 20, dev_prehash + gpb * BPT * 20, dev_out + gpb * BPT * 32);
		}
	}
	cudaStatus = hipGetLastError();
	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(out, dev_out, 32 * message_num * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(dev_TX);
	hipFree(dev_bid);
	hipFree(dev_prehash);
	hipFree(dev_out);
}