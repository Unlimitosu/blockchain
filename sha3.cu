#include "kernel.cuh"

#ifdef __cplusplus
extern "C"{
#endif

#define TXSIZE 1024*1024
#define BLOCKSIZE TXSIZE + 40
#define DIGESTSIZE 256 / 8

extern int end_offset;

typedef struct {
	uint8_t blocknum[20];
	uint8_t prevhashval[20];
	uint8_t transaction[TXSIZE];
}Block;

//! Big number addition
void addone_bignum(uint8_t* arr, size_t size) {
	for (size_t i = 0; i < size; i++) {
		if (arr[i] == 0xff) 
			arr[i] = 0;
		else {
			arr[i] += 1;
			break;
		}
	}
}

//! Print entire chain
void print_chain(Block* chain, size_t n) {
	for (int i = 0; i < n; i++) {
		printf("BlockID: ");
		for (int j = 0; j < 20; j++) {
			printf("%02x ", chain[i].blocknum[j]);
		}printf("\n");

		printf("PrevHashval: ");
		for (int j = 0; j < 20; j++) {
			printf("%02x ", chain[i].prevhashval[j]);
		}printf("\n");

		printf("Transaction:\n");
		for (int j = 0; j < 8; j++) {
			for (int k = 0; k < 128; k++) {
				//printf("%02x ", chain[i].transaction[128 * j + k]);
				//if (k==63) printf("\n");
			}
			//printf("\n");
		}printf("\n");
	}printf("\n");
}

//! print a keccack state
void print_keccackstate(uint8_t* state) {
	for (int i = 0; i < 200; i++) {
		printf("%02x ", state[i]);
		if ((i + 1)% 31 == 0) printf("\n");
	}printf("\n\n");
}

//! Create n block chains
Block* create_chain(size_t n) {
	Block* chain = (Block*)calloc(n, sizeof(Block));
	assert(chain != NULL);

	for (size_t i = 1; i < n; i++) {
		for (int j = 0; j < 20; j++) 
			chain[i].blocknum[j] = chain[i - 1].blocknum[j];
		addone_bignum(chain[i].blocknum, 20);
	}
	for (int i = 0; i < n; i++) 
		for (int j = 0; j < TXSIZE; j++) 
			chain[i].transaction[j] = rand() & 0xff;

	return chain;
}

//! Copy transaction data to txarr
void tx_info(Block* chain, size_t n, uint8_t* txarr) {
	for (int i = 0; i < n; i++) 
		for (int j = 0; j < TXSIZE; j++) 
			txarr[i * TXSIZE + j] = chain[i].transaction[j];
}

//! Copy blockID data to blockid
void blocknum_info(Block* chain, size_t n, uint8_t* blockid) {
	for (int i = 0; i < n; i++) 
		for (int j = 0; j < 20; j++) 
			blockid[i * 20 + j] = chain[i].blocknum[j];
}

//! Copy previous hash value to prehash
void prehash_info(Block* chain, size_t n, uint8_t* prehash) {
	for (int i = 0; i < n; i++) 
		for (int j = 0; j < 20; j++) 
			prehash[i * 20 + j] = chain[i].prevhashval[j];
}

//! Calculate hash value using GPU
void sha3_hash_cuda(Block* chain, size_t n, uint8_t* keccack_state) {
	uint8_t* txarr  = (uint8_t*)calloc(n * TXSIZE, sizeof(uint8_t));
	uint8_t* remain = (uint8_t*)calloc(40, sizeof(uint8_t));
	uint8_t tmp[32] = { 0, };

	assert(txarr != NULL);
	assert(remain != NULL);

	tx_info(chain, n, txarr);

	//! hash transaction data
	sha3_cpu_to_gpu(txarr, keccack_state, n);
	
	//! hash remaining data(blockID, prehash)
	for (int i = 0; i < n; i++) {
		//! copy the remaining data to remain
		memcpy(remain, chain[i].blocknum, 20);
		memcpy(remain + 20, chain[i].prevhashval, 20);

		//! use sha3 with end_offset = 16
		//! because there are already XOR-ed the 16-bytes of lsb of the transaction
		sha3_endoffset16(remain, 40, tmp, keccack_state + (200 * i));

		//! squeeze the final hash value
		memset(tmp, 0x00, 32);
		sha3_final(tmp, keccack_state + 200 * i);
		
		//! copy the last 160-bits of the hash value into the prevhashval
		memcpy(chain[i + 1].prevhashval, tmp + 12, 20);
	}
	free(txarr);
	free(remain);
}

//! hash test function
void hashtest() {
	int n = 10;

	uint8_t* blockinfo = (uint8_t*)calloc(BLOCKSIZE, sizeof(uint8_t));
	uint8_t hashval[DIGESTSIZE] = { 0, };

	uint8_t* keccack_state  = (uint8_t*)calloc(200 * n, sizeof(uint8_t));
	uint8_t* keccack_state2 = (uint8_t*)calloc(200, sizeof(uint8_t));
	assert(keccack_state != NULL);
	assert(keccack_state2 != NULL);

	Block* chain = create_chain(n);
	Block* chain2 = (Block*)calloc(n, sizeof(Block));
	assert(chain2 != NULL);

	memcpy(chain2, chain, sizeof(Block) * n);

	sha3_hash_cuda(chain, n, keccack_state);

	for (int blockid = 0; blockid < n - 1; blockid++) {
		memset(keccack_state2, 0x00, 200);

		for (int i = 0; i < TXSIZE; i++)
			blockinfo[i] = chain[blockid].transaction[i];
		for (int i = 0; i < 20; i++)
			blockinfo[i + TXSIZE] = chain[blockid].blocknum[i];
		for (int i = 0; i < 20; i++)
			blockinfo[i + TXSIZE + 20] = chain[blockid].prevhashval[i];

		sha3(blockinfo, TXSIZE, hashval, keccack_state2);
		sha3_update(blockinfo + TXSIZE, 40, keccack_state2);
		sha3_final(hashval, keccack_state2);

		memcpy(chain2[blockid + 1].prevhashval, hashval + 12, 20);
	}

	free(blockinfo);
	free(keccack_state);
	free(keccack_state2);
	free(chain);
	free(chain2);
}

//! verify the trasaction values linearly
//! use CUDA C for the performance
void verify_transaction(Block* chain, size_t n) {
	uint8_t* txarr   = (uint8_t*)calloc(n * TXSIZE, sizeof(uint8_t));
	uint8_t* bid	 = (uint8_t*)calloc(20 * n, sizeof(uint8_t));
	uint8_t* prehash = (uint8_t*)calloc(20 * n, sizeof(uint8_t));
	uint8_t* out	 = (uint8_t*)calloc(32 * n, sizeof(uint8_t));

	//! copy the data in the chain
	tx_info(chain, n, txarr);
	prehash_info(chain, n, prehash);
	blocknum_info(chain, n, bid);

	//! run CUDA C code to get hash values
	sha3_verify_cpu_to_gpu(txarr, bid, prehash, out, n);

	//! verify the hash values
	//! print ERROR if the hash value is not the same
	for (int i = 1; i < n; i++) {
		if (memcmp(chain[i].prevhashval, out + (32 * (i - 1) + 12), 20) != 0) {
			printf("%d-th chain TX ERROR\n", i - 1);
			for (int j = 0; j < 20; j++)
				printf("%02X ", chain[i].prevhashval[j]);
			printf("\n");

			for (int j = 0; j < 20; j++)
				printf("%02X ", *(out + (32 * (i - 1) + 12 + j)));
			printf("\n");

			goto END;
		}
	}
	printf("ALL BLOCK SUCCESS\n");

END:
	free(txarr);
	free(bid);
	free(prehash);
	free(out);
}

//! test function
void test() {
	int n = 10;

	uint8_t* keccack_state = (uint8_t*)calloc(200 * n, sizeof(uint8_t));
	assert(keccack_state != NULL);

	Block* chain = create_chain(n);

	sha3_hash_cuda(chain, n, keccack_state);

	verify_transaction(chain, n);

	//! modify the random index of block and transaction then verify
	int blockidx, txidx;
	blockidx = rand() % n;
	txidx = rand() % TXSIZE;

	printf("modifed block index: %d\n", blockidx);
	printf("modifed transaction index: %d\n\n", txidx);

	chain[blockidx].transaction[txidx]++;

	verify_transaction(chain, n);

	free(keccack_state);
	free(chain);
}

int main() {
	srand(time(NULL));
	test();
	return 0;
}

#ifdef __cplusplus
}
#endif